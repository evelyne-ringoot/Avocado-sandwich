#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <map>
#include <random>
#include <tuple>
#include <utility>
#include <vector>
#include <string>
#include <sstream>
#include <cmath>
#include <hipsolver.h>
#include <hiprand.h>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>

// Error checking macro for CUDA calls
#define CUDA_CHECK(call)                                                       \
    do {                                                                       \
        hipError_t error = call;                                              \
        if (error != hipSuccess) {                                            \
            std::cerr << "CUDA error: " << hipGetErrorString(error) << " at " \
                      << __FILE__ << ":" << __LINE__ << std::endl;             \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

// Error checking macro for cuSOLVER calls
#define CUSOLVER_CHECK(call)                                                   \
    do {                                                                       \
        hipsolverStatus_t status = call;                                        \
        if (status != HIPSOLVER_STATUS_SUCCESS) {                               \
            std::cerr << "cuSOLVER error: " << status << " at "                \
                      << __FILE__ << ":" << __LINE__ << std::endl;             \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

// Error checking macro for cuRAND calls
#define CURAND_CHECK(call)                                                     \
    do {                                                                       \
        hiprandStatus_t status = call;                                          \
        if (status != HIPRAND_STATUS_SUCCESS) {                                 \
            std::cerr << "cuRAND error: " << status << " at "                  \
                      << __FILE__ << ":" << __LINE__ << std::endl;             \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

//references
//https://github.com/accelerated-computing-class/lab6
//https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/gesvd/cusolver_gesvd_example.cu
//https://github.com/ROCm/rocm-examples/tree/f9d4e5e78325c36b319d91ec37c6410b2b6e12fb/Libraries/hipSOLVER/gesvd

constexpr int32_t __host__ __device__ ceil_div_static(int32_t a, int32_t b) { return (a + b - 1) / b; }


template <typename Reset, typename F>
double
benchmark_ms(double target_time_ms, int32_t num_iters_inner, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    int k=0;
    while (elapsed_ms < target_time_ms || k<2) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t i = 0; i < num_iters_inner; ++i) {
            f();
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms / num_iters_inner);
        k++;
    }
    return best_time_ms;
}

struct BenchmarkConfig {
    int32_t size_in;
};

enum class Phase {
    TEST,
    WARMUP,
    BENCHMARK,
};

void run_config( Phase phase,
    BenchmarkConfig const &config) {
    auto size_in = config.size_in;

    if (phase==Phase::BENCHMARK){
        printf("  %6d ", size_in);
    }else{
        printf("  %6d \n", size_in);
    }
 
    hiprandGenerator_t curandGen;
    CURAND_CHECK(hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(curandGen, 12345ULL));

    double *a_gpu;
    double *svdout;
    CUDA_CHECK(hipMalloc(&a_gpu, size_in * size_in * sizeof(double)));
    CUDA_CHECK(hipMalloc(&svdout, size_in * sizeof(double)));
    
    hipsolverHandle_t cusolverH = nullptr;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    int *d_info = nullptr;
    CUDA_CHECK(hipMalloc(&d_info, sizeof(int)));
    int lwork = 0;
    CUSOLVER_CHECK(hipsolverDnDgesvd_bufferSize( cusolverH,  size_in,    size_in,   &lwork   ));
    double *d_work = nullptr;
    CUDA_CHECK(hipMalloc(&d_work, lwork * sizeof(double)));

    double elapsed_ms = benchmark_ms(
        200.0,
        2,
        [&]() {
            CURAND_CHECK(hiprandGenerateUniformDouble(curandGen, a_gpu, size_in*size_in)); 
        },
        [&]() {
            CUSOLVER_CHECK(hipsolverDnDgesvd(
                cusolverH,  'N',  'N',  size_in,   size_in,  a_gpu, size_in, svdout, nullptr, 
                size_in,   nullptr,  size_in,   d_work,  lwork,   nullptr,  d_info  ));
        });

    CUDA_CHECK(hipFree(a_gpu));
    CUDA_CHECK(hipFree(svdout));
    CUDA_CHECK(hipFree(d_work));
    CUDA_CHECK(hipFree(d_info));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CURAND_CHECK(hiprandDestroyGenerator(curandGen));

    if (phase==Phase::BENCHMARK){
        printf("  %8.03f \n", elapsed_ms);
    }
}

void run_all_configs(
    Phase phase,
    std::vector<BenchmarkConfig> const &configs) {
    if (phase == Phase::WARMUP) {
        printf("warmup\n\n");
    }else {
        printf("\n\n");
        printf(
            "  %-6s  %-9s \n",
            "size_i",
            "time (ms)");
        printf(
            "  %-6s  %-9s  \n",
            "------",
            "---------");
    }
    for (auto const &config : configs) {
        run_config( phase, config);
    }
    printf("\n");
}



int main(int argc, char **argv) {
    std::string test_data_dir = ".";
    std::vector<BenchmarkConfig> configs_test;
    
    if (argc==0){
        configs_test = std::vector<BenchmarkConfig>{
            {{64},{128},{256},{512},{1024},{2048}, {4096}},
        };
    }else(
        int n = std::stoi(argv[1]);
        configs_test = std::vector<BenchmarkConfig>{
            {n},
        };
    )
    

    run_all_configs(Phase::WARMUP,  configs_test);
    run_all_configs(Phase::BENCHMARK, configs_test);

    return 0;
}


