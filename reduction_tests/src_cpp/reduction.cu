#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <map>
#include <random>
#include <tuple>
#include <utility>
#include <vector>
#include <string>
#include <sstream>

constexpr int32_t __host__ __device__ ceil_div_static(int32_t a, int32_t b) { return (a + b - 1) / b; }


void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

__device__ inline void cp_async4(void *smem_ptr, const void *glob_ptr) {
    const int BYTES = 16;
    uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    asm volatile(
        "{\n"
        "   cp.async.cg.shared.global [%0], [%1], %2;\n"
        "}\n" ::"r"(smem),
        "l"(glob_ptr),
        "n"(BYTES));
}
__device__ inline void cp_async1(void *smem_ptr, const void *glob_ptr) {
    const int BYTES = 4;
    uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    asm volatile(
        "{\n"
        "   cp.async.ca.shared.global [%0], [%1], %2;\n"
        "}\n" ::"r"(smem),
        "l"(glob_ptr),
        "n"(BYTES));
}

__device__ __forceinline__ void async_memcpy_waitall() {
    asm volatile("cp.async.wait_all;\n" ::);
}

////////////////////////////////////////////////////////////////////////////////




#define elperthread 128
#define numthreads 1
#define numpar 32



__global__ void reduction( 
    int size_in, int numsamples,
    float *input) {
    int g = blockIdx.x *blockDim.x + threadIdx.x;

    int startidx=g*size_in;
    if (g<numsamples){
        float res=input[startidx];

        for (int k=1;k<elperthread;k++){
            res+=input[startidx+k];
        }
        input[startidx]=res;
    }
    
}

void run_reduction(
    uint32_t size_i,
    uint32_t size_j,
    float *data /* pointer to GPU memory */
) {
    
    reduction<<<ceil_div_static(size_j,numpar), numthreads*numpar>>>(size_i, size_j, data);
}

////////////////////////////////////////////////////////////////////////////////

void print_matrix(int32_t n_row, int32_t n_col, std::vector<float> const &matrix) {
    for (int32_t i = 0; i < n_row; i++) {
        printf("    ");
        for (int32_t j = 0; j < n_col; j++) {
            printf("%10.5f ", matrix.at(i * n_col + j));
        }
        printf("\n");
    }
}

std::vector<float> read_data(std::string const &path, int32_t size) {
    std::ifstream file(path, std::ios::binary);
    std::vector<float> data(size);
    file.read(reinterpret_cast<char *>(data.data()), data.size() * sizeof(float));
    if (file.fail()) {
        std::cerr << "Failed to read " << path << std::endl;
        std::abort();
    }
    return data;
}

template <typename Reset, typename F>
double
benchmark_ms(double target_time_ms, int32_t num_iters_inner, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    int k=0;
    while (elapsed_ms < target_time_ms || k<2) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t i = 0; i < num_iters_inner; ++i) {
            f();
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms / num_iters_inner);
        k++;
    }
    return best_time_ms;
}

struct BenchmarkConfig {
    int32_t size_i;
    int32_t size_j;
};

struct TestData {
    std::map<std::tuple<int32_t, int32_t>, std::vector<float>> a;
    std::map<std::tuple<int32_t, int32_t>, std::vector<float>> ref;
};

TestData read_test_data(
    std::string const &test_data_dir,
    std::vector<BenchmarkConfig> const &configs) {
    auto data = TestData{};
    for (auto const &config : configs) {
        auto size_i = config.size_i;
        auto size_j = config.size_j;
        auto path_prefix = test_data_dir + "/";

        if (data.a.find({size_i, size_j}) == data.a.end()) {
            data.a[{size_i, size_j}] = read_data(
                path_prefix + "test_a_" + std::to_string(size_i) + "_" +
                    std::to_string(size_j) + ".bin",
                size_i * size_j);
        }

        if (data.ref.find({size_i, size_j}) == data.ref.end()) {
            data.ref[{size_i, size_j}] = read_data(
                path_prefix + "ref_a_" + std::to_string(size_i) + "_" +
                    std::to_string(size_j) + ".bin",
                size_j);
        }

    }
    return data;
}

struct BenchmarkResults {
    char const *name;
    std::map<std::tuple<int32_t, int32_t>, double> elapsed_ms;
};

enum class Phase {
    TEST,
    WARMUP,
    BENCHMARK,
};

void run_config( Phase phase,
    TestData const &data,
    BenchmarkConfig const &config,
    BenchmarkResults &results) {
    auto size_i = config.size_i;
    auto size_j = config.size_j;

    auto const &a = data.a.at({size_i, size_j});
    auto const &ref = data.ref.at({size_i, size_j});

    float *a_gpu;
    CUDA_CHECK(hipMalloc(&a_gpu, size_i * size_j * sizeof(float)));

    CUDA_CHECK(hipMemcpy(
        a_gpu,
        a.data(),
        size_i * size_j * sizeof(float),
        hipMemcpyHostToDevice));

    if (phase==Phase::BENCHMARK){
        printf("  %6d  %6d  ", size_i, size_j);
    }else{
        printf("  %6d  %6d \n", size_i, size_j);
    }
    run_reduction(size_i,size_j,   a_gpu);

    std::vector<float> c_out_host(size_i * size_j);
    CUDA_CHECK(hipMemcpy(
        c_out_host.data(),
        a_gpu,
        size_i * size_j * sizeof(float),
        hipMemcpyDeviceToHost));

    double mse = 0.0;
    double ref_mean_square = 0.0;
    for (int32_t i = 0; i < size_j; ++i) {
            float diff = abs(c_out_host[i * size_i ]) - abs(ref[i ]);
            mse += diff * diff;
            ref_mean_square += abs(ref[i ]) * abs(ref[i]);
    }
    float rmse = std::sqrt(mse);
    float rel_rmse = rmse; // std::sqrt(ref_mean_square);
    if (phase==Phase::BENCHMARK){
        printf("  %8.02e", rel_rmse);
    }
    

    double target_time_ms = 200.0;
    double elapsed_ms = benchmark_ms(
        target_time_ms,
        20,
        [&]() {
        },
        [&]() {
            run_reduction(size_i,size_j,   a_gpu);
        });

    results.elapsed_ms[{size_i, size_j}] = elapsed_ms;
    if (phase==Phase::BENCHMARK){
        printf("  %8.03f \n", elapsed_ms);
    }

    CUDA_CHECK(hipFree(a_gpu));
}


BenchmarkResults run_all_configs(
    Phase phase,
    TestData const &data,
    std::vector<BenchmarkConfig> const &configs) {
    auto results = BenchmarkResults{"reduction"};
    if (phase == Phase::WARMUP) {
        printf("warmup\n\n");
    }else {
        printf("\n\n");
        printf(
            "  %-6s  %-6s   %-8s  %-9s \n",
            "size_i",
            "size_j",
            "RRMSE",
            "time (ms)");
        printf(
            "  %-6s  %-6s  %-8s  %-9s  \n",
            "------",
            "------",
            "--------",
            "---------");
    }
    for (auto const &config : configs) {
        run_config( phase, data, config, results);
    }
    printf("\n");
    return results;
}



std::vector<BenchmarkResults> run_all_impls(
    Phase phase,
    TestData const &data,
    std::vector<BenchmarkConfig> const &configs) {
    auto results = std::vector<BenchmarkResults>{};
    results.push_back(run_all_configs(phase, data, configs));
    return results;
}



int main(int argc, char **argv) {
    std::string test_data_dir = ".";

    auto configs_test = std::vector<BenchmarkConfig>{
        {{128,32},{128,128},{128,512},{128,2048},{128,1024*8},{128,32768},{128,32768*4}},
    };

    auto data = read_test_data(test_data_dir, configs_test);
    run_all_impls(Phase::WARMUP, data, configs_test);
    run_all_impls(Phase::BENCHMARK, data, configs_test);
  

    return 0;
}
