#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <map>
#include <random>
#include <tuple>
#include <utility>
#include <vector>
#include <string>
#include <sstream>

constexpr int32_t __host__ __device__ ceil_div_static(int32_t a, int32_t b) { return (a + b - 1) / b; }


void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

__device__ inline void cp_async4(void *smem_ptr, const void *glob_ptr) {
    const int BYTES = 16;
    uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    asm volatile(
        "{\n"
        "   cp.async.cg.shared.global [%0], [%1], %2;\n"
        "}\n" ::"r"(smem),
        "l"(glob_ptr),
        "n"(BYTES));
}
__device__ inline void cp_async1(void *smem_ptr, const void *glob_ptr) {
    const int BYTES = 4;
    uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    asm volatile(
        "{\n"
        "   cp.async.ca.shared.global [%0], [%1], %2;\n"
        "}\n" ::"r"(smem),
        "l"(glob_ptr),
        "n"(BYTES));
}

__device__ __forceinline__ void async_memcpy_waitall() {
    asm volatile("cp.async.wait_all;\n" ::);
}

////////////////////////////////////////////////////////////////////////////////




#define numparx 8
#define numpary 8


__global__ void naive_matmul( 
    int size_in,
    float *inputa, float *inputb, float *output ) {
    int g = blockIdx.x *blockDim.x + threadIdx.x;
    int h = blockIdx.y *blockDim.y + threadIdx.y;

    if (g<size_in && h<size_in){
        float res=0.0;

        for (int k=0;k<size_in;k++){
            res+=inputa[g+size_in*k]*inputb[k+size_in*h];
        }
        output[g+size_in*h]=res;
    }
}

void run_naive_matmul(
    int size_in,
    float *inputa, float *inputb, float *output  /* pointer to GPU memory */
) {
    dim3 dimBlock(numparx, numpary);
    dim3 dimGrid(ceil_div_static(size_in,numparx), ceil_div_static(size_in,numpary));
    naive_matmul<<<dimGrid,dimBlock>>>(size_in, inputa, inputb, output);
}

////////////////////////////////////////////////////////////////////////////////

void print_matrix(int32_t n_row, int32_t n_col, std::vector<float> const &matrix) {
    for (int32_t i = 0; i < n_row; i++) {
        printf("    ");
        for (int32_t j = 0; j < n_col; j++) {
            printf("%10.5f ", matrix.at(i * n_col + j));
        }
        printf("\n");
    }
}

std::vector<float> read_data(std::string const &path, int32_t size) {
    std::ifstream file(path, std::ios::binary);
    std::vector<float> data(size);
    file.read(reinterpret_cast<char *>(data.data()), data.size() * sizeof(float));
    if (file.fail()) {
        std::cerr << "Failed to read " << path << std::endl;
        std::abort();
    }
    return data;
}

template <typename Reset, typename F>
double
benchmark_ms(double target_time_ms, int32_t num_iters_inner, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    int k=0;
    while (elapsed_ms < target_time_ms || k<2) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t i = 0; i < num_iters_inner; ++i) {
            f();
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms / num_iters_inner);
        k++;
    }
    return best_time_ms;
}

struct BenchmarkConfig {
    int32_t size_i;
};

struct TestData {
    std::map<std::tuple<int32_t>, std::vector<float>> a;
    std::map<std::tuple<int32_t>, std::vector<float>> b;
    std::map<std::tuple<int32_t>, std::vector<float>> ref;
};

TestData read_test_data(
    std::string const &test_data_dir,
    std::vector<BenchmarkConfig> const &configs) {
    auto data = TestData{};
    for (auto const &config : configs) {
        auto size_i = config.size_i;
        auto path_prefix = test_data_dir + "/";

        if (data.a.find({size_i}) == data.a.end()) {
            data.a[{size_i}] = read_data(
                path_prefix + "test_a_" + std::to_string(size_i) + "_" +
                    std::to_string(size_i) + ".bin",
                size_i * size_i);
        }

        if (data.b.find({size_i}) == data.b.end()) {
            data.b[{size_i}] = read_data(
                path_prefix + "test_b_" + std::to_string(size_i) + "_" +
                    std::to_string(size_i) + ".bin",
                size_i * size_i);
        }

        if (data.ref.find({size_i}) == data.ref.end()) {
            data.ref[{size_i}] = read_data(
                path_prefix + "ref_c_" + std::to_string(size_i) + "_" +
                    std::to_string(size_i) + ".bin",
                size_i*size_i);
        }

    }
    return data;
}

struct BenchmarkResults {
    char const *name;
    std::map<std::tuple<int32_t>, double> elapsed_ms;
};

enum class Phase {
    TEST,
    WARMUP,
    BENCHMARK,
};

void run_config( Phase phase,
    TestData const &data,
    BenchmarkConfig const &config,
    BenchmarkResults &results) {
    auto size_i = config.size_i;

    auto const &a = data.a.at({size_i});
    auto const &b = data.b.at({size_i});
    auto const &ref = data.ref.at({size_i});

    float *a_gpu;
    CUDA_CHECK(hipMalloc(&a_gpu, size_i * size_i * sizeof(float)));

    CUDA_CHECK(hipMemcpy(
        a_gpu,
        a.data(),
        size_i * size_i * sizeof(float),
        hipMemcpyHostToDevice));

    float *b_gpu;
    CUDA_CHECK(hipMalloc(&b_gpu, size_i * size_i * sizeof(float)));

    CUDA_CHECK(hipMemcpy(
        b_gpu,
        b.data(),
        size_i * size_i * sizeof(float),
        hipMemcpyHostToDevice));

            float *c_gpu;
    CUDA_CHECK(hipMalloc(&c_gpu, size_i * size_i * sizeof(float)));


    if (phase==Phase::BENCHMARK){
        printf("  %6d ", size_i);
    }else{
        printf("  %6d \n", size_i);
    }
    run_naive_matmul(size_i,   a_gpu, b_gpu, c_gpu);
    
    std::vector<float> c_out_host(size_i * size_i);
    CUDA_CHECK(hipMemcpy(
        c_out_host.data(),
        c_gpu,
        size_i * size_i * sizeof(float),
        hipMemcpyDeviceToHost));
    
    double mse = 0.0;
    double ref_mean_square = 0.0;
    for (int32_t i = 0; i < size_i*size_i; ++i) {
            float diff = abs(c_out_host[i]) - abs(ref[i ]);
            mse += diff * diff;
            ref_mean_square += abs(ref[i ]) * abs(ref[i]);
    }
    float rmse = std::sqrt(mse);
    float rel_rmse = rmse/std::sqrt(size_i); // std::sqrt(ref_mean_square);
    if (phase==Phase::BENCHMARK){
        printf("  %8.02e", rel_rmse);
    }
    

    double target_time_ms = 200.0;
    double elapsed_ms = benchmark_ms(
        target_time_ms,
        2,
        [&]() {
        },
        [&]() {
            run_naive_matmul(size_i,   a_gpu, b_gpu, c_gpu);
        });

    results.elapsed_ms[{size_i}] = elapsed_ms;
    if (phase==Phase::BENCHMARK){
        printf("  %8.03f \n", elapsed_ms);
    }

    CUDA_CHECK(hipFree(a_gpu));
    CUDA_CHECK(hipFree(b_gpu));
    CUDA_CHECK(hipFree(c_gpu));
}


BenchmarkResults run_all_configs(
    Phase phase,
    TestData const &data,
    std::vector<BenchmarkConfig> const &configs) {
    auto results = BenchmarkResults{"reduction"};
    if (phase == Phase::WARMUP) {
        printf("warmup\n\n");
    }else {
        printf("\n\n");
        printf(
            "  %-6s  %-8s  %-9s \n",
            "size_i",
            "RRMSE",
            "time (ms)");
        printf(
            "  %-6s  %-8s  %-9s  \n",
            "------",
            "--------",
            "---------");
    }
    for (auto const &config : configs) {
        run_config( phase, data, config, results);
    }
    printf("\n");
    return results;
}



std::vector<BenchmarkResults> run_all_impls(
    Phase phase,
    TestData const &data,
    std::vector<BenchmarkConfig> const &configs) {
    auto results = std::vector<BenchmarkResults>{};
    results.push_back(run_all_configs(phase, data, configs));
    return results;
}



int main(int argc, char **argv) {
    std::string test_data_dir = ".";

    auto configs_test = std::vector<BenchmarkConfig>{
        {{32},{64},{128},{256},{512},{1024},{2048}},
    };

    auto data = read_test_data(test_data_dir, configs_test);
    run_all_impls(Phase::WARMUP, data, configs_test);
    run_all_impls(Phase::BENCHMARK, data, configs_test);
  

    return 0;
}
